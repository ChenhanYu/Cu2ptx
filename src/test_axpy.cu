#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void daxpy_asm_1_1(
  double*       __restrict__ y,
  double                     alpha,
  const double* __restrict__ x )
{
  /*
  for ( int i = 0; i < 8; i++) y[ i ] += alpha * x[ i ];
  asm volatile (
    "\n\t"
    "\n\t"
    :
    "+l"( y )
    :
    "d"(alpha),
    "l"(x)
  );
  */
  #include "axpy.asm"
}

int main()
{
  double y[8] = { 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0 };
  double x[8] = { 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0 };
  double alpha = -1.2;

  double *y_d, *x_d;

  hipMalloc( ( void** ) &y_d, sizeof(double) * 8 );
  hipMalloc( ( void** ) &x_d, sizeof(double) * 8 );

  hipMemcpy( y_d, y, sizeof(double) * 8, hipMemcpyHostToDevice );
  hipMemcpy( x_d, x, sizeof(double) * 8, hipMemcpyHostToDevice );

  daxpy_asm_1_1<<< 1, 1 >>>( y_d, alpha, x_d );

  hipMemcpy( y, y_d, sizeof(double) * 8, hipMemcpyDeviceToHost );

  printf("%4.2f, %4.2f, %4.2f, %4.2f\n", y[0], y[1], y[2], y[3]);

  return 0;
}
